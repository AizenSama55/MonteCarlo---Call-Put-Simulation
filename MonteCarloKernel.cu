
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

extern "C" __global__ void MonteCarloKernel(double currentPrice, double meanReturn, double volatility, int days, double* finalPrices)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    double price = currentPrice;
    double randomShock;

    // Initialize the random state
    hiprandState state;
    hiprand_init(1234, idx, 0, &state);

    for (int t = 1; t < days; t++)
    {
        randomShock = meanReturn + hiprand_normal(&state) * volatility;
        price = price * (1 + randomShock);
    }
    finalPrices[idx] = price;
}
